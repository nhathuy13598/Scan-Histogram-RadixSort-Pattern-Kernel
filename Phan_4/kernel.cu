#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>

#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

// Sequential radix sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
void sortByHost(const uint32_t * in, int n,
	uint32_t * out,
	int nBits)
{
	int nBins = 1 << nBits; // 2^nBits
	int * hist = (int *)malloc(nBins * sizeof(int));
	int * histScan = (int *)malloc(nBins * sizeof(int));

	// In each counting sort, we sort data in "src" and write result to "dst"
	// Then, we swap these 2 pointers and go to the next counting sort
	// At first, we assign "src = in" and "dest = out"
	// However, the data pointed by "in" is read-only 
	// --> we create a copy of this data and assign "src" to the address of this copy
	uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
	memcpy(src, in, n * sizeof(uint32_t));
	uint32_t * originalSrc = src; // Use originalSrc to free memory later
	uint32_t * dst = out;

	// Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
	// (Each digit consists of nBits bits)
	// In each loop, sort elements according to the current digit 
	// (using STABLE counting sort)
	for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
	{
		// TODO: Compute "hist" of the current digit
		memset(hist, 0, nBins * sizeof(int));
		for (int i = 0; i < n; i++) {
			int bin = (src[i] >> bit) & (nBins - 1);
			hist[bin]++;
		}
		// TODO: Scan "hist" (exclusively) and save the result to "histScan"
		histScan[0] = 0;
		for (int i = 1; i < nBins; i++) {
			histScan[i] = histScan[i - 1] + hist[i - 1];
		}
		// TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
		for (int i = 0; i < n; i++) {
			int bin = (src[i] >> bit) & (nBins - 1);
			dst[histScan[bin]] = src[i];
			histScan[bin]++;
		}
		// TODO: Swap "src" and "dst"
		uint32_t *temp = src;
		src = dst;
		dst = temp;
	}

	// TODO: Copy result to "out"
	memcpy(out, src, n * sizeof(uint32_t));

	// Free memories
	free(hist);
	free(histScan);
	free(originalSrc);
}

// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
__global__ void computeHistKernel1(uint32_t * in, int n, int * hist, int bit, int nBins)
{
	// TODO
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		int bin = (in[i] >> bit) & (nBins - 1);
		atomicAdd(&hist[bin], 1);
	}
}

__global__ void scanBlkKernel(int * in, int n, int * out, int * blkSums)
{
	// TODO
	extern __shared__ uint32_t s_data[];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	s_data[threadIdx.x] = (i < n) ? in[i] : 0;
	__syncthreads();

	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		int temp = 0;
		if (threadIdx.x >= stride) {
			temp = s_data[threadIdx.x - stride];
		}
		__syncthreads();
		if (threadIdx.x >= stride) {
			s_data[threadIdx.x] += temp;
		}
		__syncthreads();
	}

	out[i] = s_data[threadIdx.x];


	if (blkSums != NULL && threadIdx.x == 0) {
		blkSums[blockIdx.x] = s_data[blockDim.x - 1];
	}
}

// TODO: You can define necessary functions here
__global__ void scanSumKernel(int *in, int *blkSums, int n) {
	if (blockIdx.x >= 1) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		in[i] += blkSums[blockIdx.x - 1];
	}
}

void sortByDevice(const uint32_t * in, int n,
	uint32_t * out,
	int nBits, int * blockSizes)
{
	// TODO
	int nBins = 1 << nBits; // 2^nBits

	// Tao mang d_hist, d_histScan va cap phat bo nho
	int *hist = (int *)malloc(nBins * sizeof(int));
	int *histScan = (int *)malloc(nBins * sizeof(int));
	int *d_hist; CHECK(hipMalloc(&d_hist, nBins * sizeof(int)));
	int *d_histScan; CHECK(hipMalloc(&d_histScan, nBins * sizeof(int)));

	// Tao mang d_in va sao chep du lieu tu in sang d_in
	uint32_t *d_in; CHECK(hipMalloc(&d_in, n * sizeof(uint32_t)));
	CHECK(hipMemcpy(d_in, in, n * sizeof(uint32_t), hipMemcpyHostToDevice));

	// Initialize gridSizeHist, gridSizeScan
	dim3 gridSizeHist((n - 1) / blockSizes[0] + 1);
	dim3 gridSizeScan((nBins - 1) / blockSizes[1] + 1);
	
	// Initialize smem_size
	int smem_size = blockSizes[1] * sizeof(int);

	// Initialize src, originalSrc, dst
	uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
	memcpy(src, in, n * sizeof(uint32_t));
	uint32_t * originalSrc = src; // Use originalSrc to free memory later
	uint32_t * dst = out;

	for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
	{
		// TODO: Compute "hist" of the current digit
		CHECK(hipMemset(d_hist, 0, nBins * sizeof(int)));
		computeHistKernel1 << <gridSizeHist, blockSizes[0] >> > (d_in, n, d_hist, bit, nBins);
		CHECK(hipGetLastError());

		// TODO: Scan "hist" (exclusively) and save the result to "histScan"
		// Khoi tao bien blkSums
		int *blkSums = (int*)malloc(gridSizeScan.x * sizeof(int));

		// Cap phat bo nho
		int *d_blkSums; CHECK(hipMalloc(&d_blkSums, gridSizeScan.x * sizeof(int)));

		// Goi ham kernel scan
		scanBlkKernel << <gridSizeScan, blockSizes[1], smem_size >> > (d_hist, nBins, d_histScan, d_blkSums);
		CHECK(hipGetLastError());

		// Chep du lieu tu device sang host
		CHECK(hipMemcpy(blkSums, d_blkSums, gridSizeScan.x * sizeof(int), hipMemcpyDeviceToHost));

		// Goi ham scan tai host cho mang blkSums
		int *scan_blkSums = (int*)malloc(gridSizeScan.x * sizeof(int));
		scan_blkSums[0] = blkSums[0];
		for (int i = 1; i < gridSizeScan.x; i++)
		{
			scan_blkSums[i] = scan_blkSums[i - 1] + blkSums[i];
		}

		// Chep du lieu tu host sang device
		CHECK(hipMemcpy(d_blkSums, scan_blkSums, gridSizeScan.x * sizeof(int), hipMemcpyHostToDevice));

		// Goi ham kernel de tinh tong
		scanSumKernel << <gridSizeScan, blockSizes[1] >> > (d_histScan, d_blkSums, nBins);
		CHECK(hipGetLastError());

		// Chep du lieu tu device sang host
		CHECK(hipMemcpy(histScan, d_histScan, nBins * sizeof(int), hipMemcpyDeviceToHost));

		// Giai phong du lieu
		CHECK(hipFree(d_blkSums));

		// Tao mang exclusive
		int * histScanTemp = (int *)malloc(nBins * sizeof(int));
		memcpy(histScanTemp, histScan, nBins * sizeof(int));
		histScan[0] = 0;
		for (int i = 1; i < nBins; i++) {
			histScan[i] = histScanTemp[i - 1];
		}

		// TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
		for (int i = 0; i < n; i++) {
			int bin = (src[i] >> bit) & (nBins - 1);
			dst[histScan[bin]] = src[i];
			histScan[bin]++;
		}
		// TODO: Swap "src" and "dst"
		uint32_t *temp = src;
		src = dst;
		dst = temp;
	}

	// TODO: Copy result to "out"
	memcpy(out, src, n * sizeof(uint32_t));

	// Free memories
	free(hist);
	free(histScan);
	free(originalSrc);
}

// Radix sort
void sort(const uint32_t * in, int n,
	uint32_t * out,
	int nBits,
	bool useDevice = false, int * blockSizes = NULL)
{
	GpuTimer timer;
	timer.Start();

	if (useDevice == false)
	{
		printf("\nRadix sort by host\n");
		sortByHost(in, n, out, nBits);
	}
	else // use device
	{
		printf("\nRadix sort by device\n");
		sortByDevice(in, n, out, nBits, blockSizes);
	}

	timer.Stop();
	printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
	for (int i = 0; i < n; i++)
	{
		if (out[i] != correctOut[i])
		{
			printf("INCORRECT :(\n");
			return;
		}
	}
	printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
	for (int i = 0; i < n; i++)
		printf("%i ", a[i]);
	printf("\n");
}

int main(int argc, char ** argv)
{
	// PRINT OUT DEVICE INFO
	printDeviceInfo();

	// SET UP INPUT SIZE
	int n = (1 << 24) + 1;
	// n = 10;
	printf("\nInput size: %d\n", n);

	// ALLOCATE MEMORIES
	size_t bytes = n * sizeof(uint32_t);
	uint32_t * in = (uint32_t *)malloc(bytes);
	uint32_t * out = (uint32_t *)malloc(bytes); // Device result
	uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

	// SET UP INPUT DATA
	for (int i = 0; i < n; i++)
		in[i] = rand();
	//printArray(in, n);

	// SET UP NBITS
	int nBits = 4; // Default
	if (argc > 1)
		nBits = atoi(argv[1]);
	printf("\nNum bits per digit: %d\n", nBits);

	// DETERMINE BLOCK SIZES
	int blockSizes[2] = { 512, 512 }; // One for histogram, one for scan
	if (argc == 4)
	{
		blockSizes[0] = atoi(argv[2]);
		blockSizes[1] = atoi(argv[3]);
	}
	printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

	// SORT BY HOST
	sort(in, n, correctOut, nBits);
	//printArray(correctOut, n);

	// SORT BY DEVICE
	sort(in, n, out, nBits, true, blockSizes);
	checkCorrectness(out, correctOut, n);

	// FREE MEMORIES 
	free(in);
	free(out);
	free(correctOut);

	return EXIT_SUCCESS;
}


